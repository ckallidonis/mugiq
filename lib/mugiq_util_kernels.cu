#include "hip/hip_runtime.h"
#include <mugiq_util_kernels.cuh>

template <typename Float>
__global__ void phaseMatrix_kernel(complex<Float> *phaseMatrix, int *momMatrix, MomProjArg *arg){

  int tid = threadIdx.x + blockIdx.x*blockDim.x;
  
  if(tid < arg->locV3){ // run through the spatial volume
    
    int lcoord[MOM_DIM_];
    int gcoord[MOM_DIM_];
    
    int a1 = tid / arg->localL[0];
    int a2 = a1 / arg->localL[1];
    lcoord[0] = tid - a1 * arg->localL[0];
    lcoord[1] = a1  - a2 * arg->localL[1];
    lcoord[2] = a2;
    
    gcoord[0] = lcoord[0] + arg->commCoord[0] * arg->localL[0];
    gcoord[1] = lcoord[1] + arg->commCoord[1] * arg->localL[1];
    gcoord[2] = lcoord[2] + arg->commCoord[2] * arg->localL[2];
    
    Float sgn = (Float) arg->FTSign;
    for(int im=0;im<arg->Nmom;im++){
      Float phase = 0.0;
      for(int id=0;id<arg->momDim;id++)
	phase += momMatrix[MOM_MATRIX_IDX(id,im)]*gcoord[id] / (Float)arg->totalL[id];
      
      phaseMatrix[tid + arg->locV3*im].x =     cos(2.0*PI*phase);
      phaseMatrix[tid + arg->locV3*im].y = sgn*sin(2.0*PI*phase);
    }
    
  }//-- tid check
  
}//--kernel

template __global__ void phaseMatrix_kernel<float> (complex<float>  *phaseMatrix, int *momMatrix, MomProjArg *arg);
template __global__ void phaseMatrix_kernel<double>(complex<double> *phaseMatrix, int *momMatrix, MomProjArg *arg);
//---------------------------------------------------------------------------



template <typename Float>
__global__ void convertIdxMomProj_kernel(complex<Float> *dataOut, const complex<Float> *dataIn, ConvertIdxArg *arg){

  int x_cb  = blockIdx.x*blockDim.x + threadIdx.x; // checkerboard site within 4d local volume
  int pty   = blockIdx.y*blockDim.y + threadIdx.y; // parity (even/odd)
  int idata = blockIdx.z*blockDim.z + threadIdx.z; // idata index
  
  if(x_cb >= arg->volumeCB) return;
  if(pty  >= arg->nParity)  return;
  
  int crd[5];
  getCoords(crd, x_cb, arg->localL, pty);
  int x = crd[0];
  int y = crd[1];
  int z = crd[2];
  int t = crd[3];

  int tid = x_cb + arg->volumeCB*pty; // full site index
  int idxFrom = tid + arg->volumeCB*arg->nParity*idata;

  int v3 = x + arg->localL[0]*y + arg->localL[0]*arg->localL[1]*z; // x + Lx*y + Lx*Ly*z
  int idxTo = v3 + arg->locV3*idata +  arg->locV3*arg->Ndata*t;

  dataOut[idxTo] = dataIn[idxFrom];  
}

template __global__ void convertIdxMomProj_kernel<float> (complex<float> *dataOut,  const complex<float> *dataIn,
							  ConvertIdxArg *arg);
template __global__ void convertIdxMomProj_kernel<double>(complex<double> *dataOut, const complex<double> *dataIn,
							  ConvertIdxArg *arg);
//---------------------------------------------------------------------------
