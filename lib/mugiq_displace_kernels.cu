#include "hip/hip_runtime.h"
#include <mugiq_displace_kernels.cuh>
/*
//- Whether a site is even (return 0) or odd (return 1)
inline static __device__ int everORodd(const int x[]){
  return (x[0] + x[1] + x[2] + x[3]) % 2;
}

template <typename Float>
inline static __device__ Link<Float> getNbrLink(Gauge<Float> &U, const int coord[], int pty,
						int dir, DisplaceSign dispSign,
						const int dim[], const int commDim[], const int nFace){

  int nbrPty = 1 - pty; //- Parity of neighboring site
  
  Link<Float> dispU;
  if (dispSign == DispSignPlus){ //- dispU_d(x) <- U_d(x) (no need to take neighbouring site)
    dispU = U(dir, linkIndex(coord, dim), pty);
  }
  else if(dispSign == DispSignMinus){ //- dispU_d(x) <- U_d^\dag(x-d)
    //- We are at the left boundary, get backward neighbouring site from the halos
    if (commDim[dir] && (coord[dir] - nFace < 0)) {
      const int ghostIdx = ghostFaceIndex<0>(coord, dim, dir, nFace);
      const Link<Float> U2 = U.Ghost(dir, ghostIdx, nbrPty);
      dispU = conj(U2);
    }
    //- Not at the boundary
    else{
      const int bwdIdx = linkIndexM1(coord, dim, dir);
      const Link<Float> U2 = U(dir, bwdIdx, nbrPty);
      dispU = conj(U2);
    }
  }
  return dispU;
}
//-------------------------------------------------------------------
//-------------------------------------------------------------------


template <typename Float>
inline static __device__ Link<Float> getNbrLinkDispExtG(Gauge<Float> &U, const int coord[], int pty,
							const int dx[],
							int dir, DisplaceSign dispSign,
							const int dimEx[], const int brd[]){
  int dx1[5] = {0,0,0,0,0};
  int c2[5]  = {0,0,0,0,0};
  for (int i=0;i<N_DIM_;i++){
    dx1[i] = dx[i];
    c2[i]  = coord[i] + brd[i];
  }

  //- For positive displacements we stay on the current site
  //- For negative displacements, move one site backwards in the "dir" direction  
  if(dispSign == DispSignMinus) dx1[dir] -= 1;
  
  int nbrPty = (everORodd(dx1) == 0) ? pty : 1 - pty; //- Parity of neighboring site (even or odd)
  
  Link<Float> dispU;
  if (dispSign == DispSignPlus)
    dispU = U(dir, linkIndexShift(c2, dx1, dimEx), nbrPty); //- dispU_d(x) <- U_d(x)
  else{
    const Link<Float> U2 = U(dir, linkIndexShift(c2, dx1, dimEx), nbrPty);  //- U2_d(x) <- U_d(x-d)
    dispU = conj(U2);                                                       //- dispU_d(x) <- U_d^\dag(x-d)
  }
  
  return dispU;
}

template <typename Float>
inline static __device__ Link<Float> getNbrLinkExtG(Gauge<Float> &U, const int coord[], int pty,
						    int dir, DisplaceSign dispSign,
						    const int dimEx[], const int brd[]){
  int dx[5] = {0,0,0,0,0};
  return getNbrLinkDispExtG<Float>(U, coord, pty, dx, dir, dispSign, dimEx, brd);
}
//-------------------------------------------------------------------
//-------------------------------------------------------------------


template <typename Float>
inline static __device__ Vector<Float> getNbrSiteVec(Fermion<Float> &F, const int coord[], int pty,
						     int dir, DisplaceSign dispSign,
						     const int dim[], const int commDim[], const int nFace){

  int nbrPty = 1 - pty; //- Parity of neighboring site
  
  Vector<Float> dispV;
  if (dispSign == DispSignPlus){ //- dispV <- F(x+d)
    //- We are at the right boundary, get forward neighbouring site from the halos
    if (commDim[dir] && (coord[dir] + nFace >= dim[dir]) ) { 
      const int ghostIdx = ghostFaceIndex<1>(coord, dim, dir, nFace);
      dispV = F.Ghost(dir, 1, ghostIdx, nbrPty);
    }
    //- Not at the boundary
    else{ 
      const int fwdIdx = linkIndexP1(coord, dim, dir);
      dispV = F(fwdIdx, nbrPty);
    }
  }
  else if(dispSign == DispSignMinus){ //- dispV <- F(x-d)
    //- We are at the left boundary, get backward neighbouring site from the halos
    if (commDim[dir] && (coord[dir] - nFace < 0)) {  
      const int ghostIdx = ghostFaceIndex<0>(coord, dim, dir, nFace);
      dispV = F.Ghost(dir, 0, ghostIdx, nbrPty);
    }
    //- Not at the boundary
    else{
      const int bwdIdx = linkIndexM1(coord, dim, dir);
      dispV = F(bwdIdx, nbrPty);
    }
  }
  return dispV;
}
//-------------------------------------------------------------------
//-------------------------------------------------------------------

*/
template <typename Float, typename Arg>
__global__ void covariantDisplacementVector_kernel(Arg *arg,
						   DisplaceDir dispDir, DisplaceSign dispSign){
  /*
  int x_cb = blockIdx.x*blockDim.x + threadIdx.x;
  int pty  = blockIdx.y*blockDim.y + threadIdx.y;
  pty = (arg->nParity == 2) ? pty : arg->parity;
  if (x_cb >= arg->volumeCB) return;
  if (pty >= arg->nParity) return;

  //- Get the local coordinates (must be 5d in case of DW fermions, not applicable here)
  int coord[5];
  getCoords(coord, x_cb, arg->dim, pty);
  coord[4] = 0;

  int dir = (int)dispDir; //- Direction of the displacement (0:x, 1:y, 2:z, 3:t)

  //- The neighbouring vector of site x, V(x+d) or V(x-d)
  Vector<Float> nbrV = getNbrSiteVec<Float>(arg->src, coord, pty, dir, dispSign, arg->dim, arg->commDim, arg->nFace);

  Link<Float> nbrU; //- Neighbouring Link, U_d(x) or U_d^\dag(x-d)
  if(arg->extendedGauge)
    nbrU = getNbrLinkExtG<Float>(arg->U, coord, pty, dir, dispSign, arg->dimEx, arg->brd);
  else
    nbrU = getNbrLink<Float>(arg->U, coord, pty, dir, dispSign, arg->dim, arg->commDim, arg->nFace);

  arg->dst(x_cb, pty) = nbrU * nbrV; // dst(x) = U_d(x) * V(x+d) || U_d^\dag(x-d) * V(x-d)
  */
}

template __global__ void covariantDisplacementVector_kernel<float, CovDispVecArg<float,QUDA_FLOAT2_FIELD_ORDER>>
(CovDispVecArg<float, QUDA_FLOAT2_FIELD_ORDER> *arg, DisplaceDir dispDir, DisplaceSign dispSign);
template __global__ void covariantDisplacementVector_kernel<float, CovDispVecArg<float,QUDA_FLOAT4_FIELD_ORDER>>
(CovDispVecArg<float, QUDA_FLOAT4_FIELD_ORDER> *arg, DisplaceDir dispDir, DisplaceSign dispSign);
template __global__ void covariantDisplacementVector_kernel<double, CovDispVecArg<double,QUDA_FLOAT2_FIELD_ORDER>>
(CovDispVecArg<double, QUDA_FLOAT2_FIELD_ORDER> *arg, DisplaceDir dispDir, DisplaceSign dispSign);
template __global__ void covariantDisplacementVector_kernel<double, CovDispVecArg<double,QUDA_FLOAT4_FIELD_ORDER>>
(CovDispVecArg<double, QUDA_FLOAT4_FIELD_ORDER> *arg, DisplaceDir dispDir, DisplaceSign dispSign);
